﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void printDeviceNames() 
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Indeks urzadzenia: %d\n", i);
        printf("Nazwa urzadzenia: %s\n\n", prop.name);
    }
}

void printDevicePropertiesById(int id) 
{
    int nDevices, nProcs;
    hipGetDeviceCount(&nDevices);
    if (id < nDevices) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, id);
        hipDeviceGetAttribute(&nProcs, hipDeviceAttributeMultiprocessorCount, id);

        printf("Nazwa urzadzenia: %s\n", prop.name);
        printf("Ilosc multiprocesorow: %d.\n", nProcs);
        printf("Kompatybilnosc obliczeniowa: %d.%d.\n\n", prop.major, prop.minor);
    }
    else 
    {
        printf("Urzadzenie o indeksie %d nie istnieje...\n", id);
    }
}

void reportGPUMemory()
{
    size_t free, total;
    int freeMem, totalMem;
    hipMemGetInfo(&free, &total);

    freeMem = static_cast<int>(free / 1048576);
    totalMem = static_cast<int>(total / 1048576);
    printf("---------- STAN PAMIECI ----------\n");
    printf("Wolna: %d MB\nCalkowita: %d MB\nUzywana: %d MB\n\n", freeMem, totalMem, totalMem - freeMem);
}

void allocAndFreeMem()
{
    reportGPUMemory();

    printf("Alokowanie 16MB danych typu char...\n\n");
    char* charData;
    hipMallocManaged((void**)&charData, 1 << 24);
    reportGPUMemory();

    printf("Alokowanie 128MB danych typu float...\n\n");
    float* floatData;
    hipMallocManaged((void**)&floatData, 1 << 27);
    reportGPUMemory();

    printf("Zwolnienie pamieci danych typu char...\n");
    hipFree(charData);
    reportGPUMemory();

    printf("Zwolnienie pamieci danych typu float...\n");
    hipFree(floatData);
    reportGPUMemory();
}

void hostAndDeviceTransfer(float *cpuTimeGlobal, float *gpuTimeGlobal, int nElements) 
{
    hipEvent_t startHostTransfer, stopHostTransfer;
    hipEvent_t startDevTransfer, stopDevTransfer;

    float cpuTime = 0, gpuTime = 0;

    int *deviceArray;
    int *hostArray = (int*)malloc(nElements * sizeof(int));
    hipMalloc((int**)&deviceArray, nElements * sizeof(int));

    hipEventCreate(&startDevTransfer);
    hipEventCreate(&stopDevTransfer);

    hipEventRecord(startDevTransfer);
    hipError_t deviceToHost = hipMemcpy(deviceArray, hostArray, nElements * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stopDevTransfer);
    hipEventSynchronize(stopDevTransfer);
    if (deviceToHost != hipSuccess) {
        printf("Nie udalo sie przekopiowac danych z GPU do CPU!\n");
    }
    else {
        printf("Kopiowanie z GPU do CPU powiodlo sie!\n");
        hipEventElapsedTime(&gpuTime, startDevTransfer, stopDevTransfer);
        printf("Czas kopiowania danych: %fs\n\n", gpuTime);
        *gpuTimeGlobal = gpuTime;
    }

    hipEventCreate(&startHostTransfer);
    hipEventCreate(&stopHostTransfer);

    hipEventRecord(startHostTransfer);
    hipError_t hostToDevice = hipMemcpy(hostArray, deviceArray, nElements * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stopHostTransfer);
    hipEventSynchronize(stopHostTransfer);
    if (hostToDevice != hipSuccess) {
        printf("Nie udalo sie przekopiowac danych z CPU do GPU!\n");
    }
    else {
        printf("Kopiowanie z CPU do GPU powiodlo sie!\n");
        hipEventElapsedTime(&cpuTime, startHostTransfer, stopHostTransfer);
        printf("Czas kopiowania danych: %fs\n\n", cpuTime);
        *cpuTimeGlobal = cpuTime;
    }
}

float computeAverageValue(float *array)
{
    float sum = 0.0;
    for (int i = 0; i < sizeof(array); i++)
    {
        sum += array[i];
    }

    return sum / sizeof(array);
}

int main()
{
    // Zadanie 1
    printDeviceNames();

    // Zadanie 2
    int id;
    printf("Wprowadz indeks urzadzenia, aby wyswietlic jego parametry: ");
    scanf("%d", &id);
    printDevicePropertiesById(id);

    // Zadanie 3
    allocAndFreeMem();

    // Zadanie 4, 5, 6
    int nElements = 1024 * 1024;
    float cpuTime[10] = { 0.0 }, gpuTime[10] = { 0.0 };
    for (int i = 0; i < 10; i++) {
        hostAndDeviceTransfer(&cpuTime[i], &gpuTime[i], nElements);
    }

    unsigned int bytes = nElements * sizeof(int);
    float avgCpySpeedCPU = (bytes * 1e-6) / computeAverageValue(cpuTime);
    float avgCpySpeedGPU = (bytes * 1e-6) / computeAverageValue(gpuTime);

    printf("\n-----------------------------------------------\n");
    printf("Rozmiar kopiowanych danych: %d MB\n", bytes / (1024 * 1024));
    printf("Srednia predkosc kopiowania dla CPU [GB/s]: %f\n", avgCpySpeedCPU);
    printf("Srednia predkosc kopiowania dla GPU [GB/s]: %f\n", avgCpySpeedGPU);

    
    return 0;
}
